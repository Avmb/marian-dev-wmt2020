#include "hip/hip_runtime.h"

#include <hipblas.h>

// clang-format off
#include "tensors/gpu/prod.h"
#include "tensors/gpu/backend.h"
#include "tensors/gpu/cuda_helpers.h"
// clang-format on

namespace marian {

namespace gpu {

static void setTensorMode(hipblasHandle_t cublasHandle) {
  static int mode = 0;  // 1: use TC; -1: do not use TC; 0: not set yet
  if (mode == 0) { // multi-thread note: this is sort-of thread-safe, since multiple threads would determine the same value
    const char* var = getenv("ENABLE_CUBLAS_TENSOR_OP_MATH_FP32");
    if (!var)
      var = "1";
    switch(var[0]) {
    case '0': mode = -1; break;
    case '1': mode =  1; break;
    default: ABORT("Invalid ENABLE_CUBLAS_TENSOR_OP_MATH_FP32={}", var);
    }
    if (mode > 0) { // try whether it can be set   --@TODO: check whether this actually works
      hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
      hipblasMath_t actual = HIPBLAS_DEFAULT_MATH;
      hipblasGetMathMode(cublasHandle, &actual);
      if (actual != HIPBLAS_TENSOR_OP_MATH) {
        LOG(info, "WARNING: TensorCores requested but not available");
        mode = -1;
      }
    }
    if (mode > 0)
      LOG(info, "16-bit TensorCores enabled for float32 matrix operations");
  }
  hipblasSetMathMode(cublasHandle, mode > 0 ? HIPBLAS_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH);
}


hipblasStatus_t cublasGemmTyped(hipblasHandle_t handle,
                               hipblasOperation_t transa, 
                               hipblasOperation_t transb,
                               int m, int n, int k,
                               const float* alpha,
                               const float* A, int lda,
                               const float* B, int ldb,
                               const float* beta,
                               float* C, int ldc) {
  return hipblasSgemm(handle, transa, transb, 
                     m, n, k, alpha, 
                     A, lda, B, ldb, beta, C, ldc);
}

hipblasStatus_t cublasGemmTyped(hipblasHandle_t handle,
                               hipblasOperation_t transa, 
                               hipblasOperation_t transb,
                               int m, int n, int k,
                               const half* alpha,
                               const half* A, int lda,
                               const half* B, int ldb,
                               const half* beta,
                               half* C, int ldc) {
  return hipblasHgemm(handle, transa, transb, 
                     m, n, k, alpha, 
                     A, lda, B, ldb, beta, C, ldc);
}

template <typename T>
void ProdTyped(marian::Tensor C,
               const marian::Tensor& A,
               const marian::Tensor& B,
               bool transA,
               bool transB,
               T beta,
               T scalar) {
  hipSetDevice(C->getDeviceId().no);
  T alpha = scalar;

  size_t m = A->shape().elements() / A->shape().back();
  size_t k = A->shape().back();
  if(transA)
    std::swap(m, k);

  size_t l = B->shape().elements() / B->shape().back();
  size_t n = B->shape().back();
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape().back();
  size_t ldb = B->shape().back();
  size_t ldc = B->shape().back();

  if(transB)
    ldc = B->shape().elements() / B->shape().back();

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  auto cublasHandle = std::static_pointer_cast<gpu::Backend>(C->getBackend())
                          ->getCublasHandle();

#if CUDA_VERSION >= 9000
  setTensorMode(cublasHandle);
  //hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
#endif
  cublasGemmTyped(cublasHandle,
                  opB,
                  opA,
                  n,
                  m,
                  k,
                  &alpha,
                  B->data<T>(),
                  ldb,
                  A->data<T>(),
                  lda,
                  &beta,
                  C->data<T>(),
                  ldc);

#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH);
#endif
}

void Prod(marian::Tensor C,
          const marian::Tensor& A,
          const marian::Tensor& B,
          bool transA,
          bool transB,
          float beta,
          float scalar) {
  if(C->type() == Type::float32) {
    ProdTyped<float>(C, A, B, transA, transB, beta, scalar);
  } else if(C->type() == Type::float16) {
    ProdTyped<half>(C, A, B, transA, transB, __float2half(beta), __float2half(scalar));
  } else {
    ABORT("Prod not implemented for type {}", C->type());
  }
}

hipblasStatus_t cublasGemmBatchedTyped(hipblasHandle_t handle,
                                      hipblasOperation_t transa, 
                                      hipblasOperation_t transb,
                                      int m, int n, int k,
                                      const float *alpha,
                                      const float *Aarray[], int lda,
                                      const float *Barray[], int ldb,
                                      const float *beta,
                                      float *Carray[], int ldc, 
                                      int batchCount) {
  return
  hipblasSgemmBatched(handle, transa, transb, 
                     m, n, k, alpha, 
                     Aarray, lda, Barray, ldb, beta,
                     Carray, ldc, batchCount);
}

hipblasStatus_t cublasGemmBatchedTyped(hipblasHandle_t handle,
                                      hipblasOperation_t transa, 
                                      hipblasOperation_t transb,
                                      int m, int n, int k,
                                      const half *alpha,
                                      const half *Aarray[], int lda,
                                      const half *Barray[], int ldb,
                                      const half *beta,
                                      half *Carray[], int ldc, 
                                      int batchCount) {
  return
  hipblasHgemmBatched(handle, transa, transb, 
                     m, n, k, alpha, 
                     Aarray, lda, Barray, ldb, beta,
                     Carray, ldc, batchCount);
}

template <typename T>
void ProdBatchedTyped(marian::Tensor C,
                      Ptr<Allocator> allocator,
                      const marian::Tensor A,
                      const marian::Tensor B,
                      bool transA,
                      bool transB,
                      T beta,
                      T scalar) {
  hipSetDevice(C->getDeviceId().no);
  T alpha = scalar;

  size_t batchA = A->shape().elements() / (A->shape()[-1] * A->shape()[-2]);
  size_t batchB = B->shape().elements() / (B->shape()[-1] * B->shape()[-2]);

  size_t m = A->shape()[-2];
  size_t k = A->shape()[-1];
  if(transA)
    std::swap(m, k);

  size_t l = B->shape()[-2];
  size_t n = B->shape()[-1];
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape()[-1];
  size_t ldb = B->shape()[-1];
  size_t ldc = B->shape()[-1];

  if(transB)
    ldc = B->shape()[-2];

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  auto cublasHandle = std::static_pointer_cast<gpu::Backend>(C->getBackend())
                          ->getCublasHandle();

  int strideA = batchA == 1 ? 0 : m * k;
  int strideB = batchB == 1 ? 0 : n * k;
  int strideC = n * m;
  int batchC = std::max(batchA, batchB);

  std::vector<const T*> aptr;
  std::vector<const T*> bptr;
  std::vector<T*> cptr;

  for(int i = 0; i < batchC; i++) {
    aptr.push_back(A->data<T>() + (i % batchA) * strideA);
    bptr.push_back(B->data<T>() + (i % batchB) * strideB);
    cptr.push_back(C->data<T>() + i * strideC);
  }

  IPtr<MemoryPiece> mp_aptr = allocator->alloc<const T*>(aptr.size());
  CudaCopy(
      aptr.data(), aptr.data() + aptr.size(), mp_aptr->data<const T*>());

  IPtr<MemoryPiece> mp_bptr = allocator->alloc<const T*>(bptr.size());
  CudaCopy(
      bptr.data(), bptr.data() + bptr.size(), mp_bptr->data<const T*>());

  IPtr<MemoryPiece> mp_cptr = allocator->alloc<T*>(cptr.size());
  CudaCopy(cptr.data(), cptr.data() + cptr.size(), mp_cptr->data<T*>());

#if CUDA_VERSION >= 9000
  setTensorMode(cublasHandle);
  //hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
#endif
  cublasGemmBatchedTyped(cublasHandle,
                         opB,
                         opA,
                         n,
                         m,
                         k,
                         &alpha,
                         mp_bptr->data<const T*>(),
                         ldb,
                         mp_aptr->data<const T*>(),
                         lda,
                         &beta,
                         mp_cptr->data<T*>(),
                         ldc,
                         batchC);
#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH);
#endif

  allocator->free(mp_aptr);
  allocator->free(mp_bptr);
  allocator->free(mp_cptr);
}

void ProdBatched(marian::Tensor C,
                 Ptr<Allocator> allocator,
                 const marian::Tensor A,
                 const marian::Tensor B,
                 bool transA,
                 bool transB,
                 float beta,
                 float scalar) {
  if(C->type() == Type::float32) {
    ProdBatchedTyped<float>(C, allocator, A, B, transA, transB, beta, scalar);
  } else if(C->type() == Type::float16) {
    ProdBatchedTyped<half>(C, allocator, A, B, transA, transB, __float2half(beta), __float2half(scalar));
  } else {
    ABORT("ProdBatched not implemented for type {}", C->type());
  }
}

}  // namespace gpu
}  // namespace marian
